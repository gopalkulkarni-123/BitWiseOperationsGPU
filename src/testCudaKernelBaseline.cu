#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void bitwiseAndKernel(const int* a, const int* b, int* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] = a[idx] & b[idx];
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int N_arr[6] = {10, 100, 1000, 10000, 100000, 1000000};  // 1 million elements

    for (int j = 0; j < 6; ++j){    
        float sum = 0.0f, avg = 0.0f;
        for (int iter = 0; iter < 11; ++iter){    

            size_t size = N_arr[j] * sizeof(int);

            // Host vectors
            int* h_a = new int[N_arr[j]];
            int* h_b = new int[N_arr[j]];
            int* h_result = new int[N_arr[j]];

            // Random bit generation
            std::mt19937 rng(std::random_device{}());
            std::uniform_int_distribution<int> bitDist(0, 1);
            for (int i = 0; i < N_arr[j]; ++i) {
                h_a[i] = bitDist(rng);
                h_b[i] = bitDist(rng);
            }

            // Device memory allocation
            int *d_a, *d_b, *d_result;
            checkCudaError(hipMalloc(&d_a, size), "Allocating d_a");
            checkCudaError(hipMalloc(&d_b, size), "Allocating d_b");
            checkCudaError(hipMalloc(&d_result, size), "Allocating d_result");

            // Copy to device
            checkCudaError(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice), "Copying h_a to d_a");
            checkCudaError(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice), "Copying h_b to d_b");

            // Kernel launch configuration
            int threadsPerBlock = 256;
            int blocksPerGrid = (N_arr[j] + threadsPerBlock - 1) / threadsPerBlock;

            // Time kernel execution
            auto start = std::chrono::high_resolution_clock::now();

            bitwiseAndKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, N_arr[j]);

            checkCudaError(hipGetLastError(), "Kernel launch");
            checkCudaError(hipDeviceSynchronize(), "Kernel sync");

            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> duration = end - start;

            //std::cout << "Kernel execution time: " << duration.count() << " ms\n";
            if (iter != 0){
                sum += duration.count();
            }

            // Copy result back
            checkCudaError(hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost), "Copying d_result to h_result");

            // Optional: Print first 10 results
            
            /*std::cout << "First 10 AND results: ";
            for (int i = 0; i < 10; ++i) {
                std::cout << h_result[i] << " ";
            }
            std::cout << "\n";*/

            // Cleanup
            hipFree(d_a);
            hipFree(d_b);
            hipFree(d_result);
            delete[] h_a;
            delete[] h_b;
            delete[] h_result;
        }
        avg = sum/10;
        std::cout << N_arr[j] << "," << avg << "ms \n";               
    }

    return 0;
}
